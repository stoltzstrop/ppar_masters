#include "hip/hip_runtime.h"
#include "Audio.h" 
#include "timing.h"
#include "timing_macros.h"
#include "constants.h"
#include <stdio.h>
#include <string.h>
#define inputfname  "Godin44100.wav"
#define outputfname "BasicRoom.wav"

// Version originally written by Craig Webb
// Updating for this project 


// there was a caching "optimisation" in the original
// it doesn't always make the code run faster !
enum BOOLEAN useOptimisation = TRUE;


// function written by Craig Webb, borrowed here 
void checkLastCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
	{
	    fprintf(stderr, "\nCuda error: %s: %s.\n", msg, hipGetErrorString( err) );
	    exit(EXIT_FAILURE);
	}                         
}

// kernel methods
__global__ void UpDateScheme(double *u,const double* __restrict__ u1);
__global__ void inout(double *u,double *out,double ins,int n);

// struct for coeffs
typedef struct
{
	double l2;
	double loss1;
	double loss2;

} coeffs_type;

// boundary conditions struct for GPU  
__constant__ coeffs_type cf_d[1];

/*
 * Main room simulation for cuda version
 */

int main(int argc, char *argv[]){

    // can also turn off the optimisation from the command line argument "off"
    if( argc > 1 )
    {
        if(strcmp(argv[1],"off") == 0)
        {
            printf("turning off optimisations...\n");
            useOptimisation = FALSE; 
        }
    }

    // Simulation parameters					      
    int NF            = numberSamples;
    double SR         = (double) numberSamples;             // Sample Rate
    double alpha      = 0.005;               // Boundary loss
    double c          = 344.0;               // Speed of sound in air (m/s)
    double k          = 1/SR;                                    
    double h          = sqrt(3.0)*c*k;
    double lambda     = c*k/h;                        

    // Set constant memory coeffs
    coeffs_type cf_h[1];
	cf_h[0].l2      = (c*c*k*k)/(h*h);
	cf_h[0].loss1   = 1.0/(1.0+lambda*alpha);
	cf_h[0].loss2   = 1.0-lambda*alpha;
     hipMemcpyToSymbol(HIP_SYMBOL(cf_d),cf_h,sizeof(coeffs_type)) ;


    char cudaStr[100];
    if(useOptimisation)
    {
        sprintf(cudaStr,"%s",CUDA_STR);
    }
    else
    {
        sprintf(cudaStr,"%s_unoptimised",CUDA_STR);
    }

    printToTimingFileName(cudaStr);

    startTime = getTime(); 
    //-------------------------------------------
    // Initialise input
    int n;
    int alength = dur;
    double *si_h = (double *)calloc(NF,sizeof(double));
    for(n=0;n<dur;n++){
      si_h[n] = 0.5*(1.0-cos(2.0*pi*n/(double)dur));
    }
    if(si_h == NULL) {
    	printf("\nFailed to open input file...\n\n"); exit(EXIT_FAILURE);
    }
    
	// Set up grid and blocks
	int Gx          = Nx/Bx;
	int Gy          = Ny/By;
	int Gz          = (Nz-2)/Bz;
	
	dim3 dimBlockInt(Bx, By, Bz);
	dim3 dimGridInt(Gx, Gy, Gz);
	dim3 dimBlockIO(1, 1, 1);
	dim3 dimGridIO(1, 1, 1);

	size_t pr_size  = sizeof(double);
	size_t mem_size = AREA*Nz*pr_size;
	double *out_d, *u_d, *u1_d, *dummy_ptr;                      
	double ins;             
    
        
        dataCopyInitStart = getTime();

	// Initialise memory on device
	 hipMalloc(&u_d, mem_size) ;       hipMemset(u_d, 0, mem_size) ;
	 hipMalloc(&u1_d, mem_size) ;      hipMemset(u1_d, 0, mem_size) ;
	 hipMalloc(&out_d, NF*pr_size) ;   hipMemset(out_d, 0, NF*pr_size) ;
	
        dataCopyInitEnd = getTime();
        dataCopyInitTotal = dataCopyInitEnd - dataCopyInitStart;

	//-------------------------------------------
	// initialise memory on host
	double *out_h  = (double *)malloc(NF*pr_size);
	double *u_h  = (double *)malloc(AREA*Nz*pr_size);
	if((out_h == NULL)){
		printf("\nout_h memory alloc failed...\n");
		exit(EXIT_FAILURE);
	}
	
        if(useOptimisation) // switch off the cache optimisation where applicable
        {
            printf("Switching on cache config\n");
	    hipFuncSetCacheConfig(reinterpret_cast<const void*>(UpDateScheme),hipFuncCachePreferL1);
	}

        kernel1Time = 0.0;
        kernel2Time = 0.0;
	dataCopyBtwTotal = 0.0;

        startKernels = getTime();

        // loop over number of timesteps 
        for(n=0;n<NF;n++)
	{
		
                startKernel1 = getTime();

                // call main room update kernel
		UpDateScheme<<<dimGridInt,dimBlockInt>>>(u_d,u1_d);
                hipDeviceSynchronize() ; 
                endKernel1 = getTime();
                kernel1Time += endKernel1-startKernel1;
	        checkLastCUDAError("1st kernel");
		
		// perform  in out
		ins = 0.0;
		if(n<alength)
                {
                  ins = si_h[n];
                }
                startKernel2 = getTime();

                // call secondary kernel to update source and receiver
                inout<<<dimGridIO,dimBlockIO>>>(u_d,out_d,ins,n);
	        hipDeviceSynchronize() ; 
                endKernel2 = getTime();
                kernel2Time += endKernel2-startKernel2;
	        checkLastCUDAError("2nd kernel");
	
                dataCopyBtwStart = getTime();

		// swap  pointers
		dummy_ptr = u1_d;
		u1_d = u_d;
		u_d = dummy_ptr;
                dataCopyBtwEnd = getTime();
                dataCopyBtwTotal += dataCopyBtwEnd - dataCopyBtwStart;
	}
	
        endKernels = getTime();
    
        kernelsTime = endKernels-startKernels;
	checkLastCUDAError("Kernel");
        hipDeviceSynchronize() ; 
    
        dataCopyBackStart = getTime();     

        // copy result back from device
        hipMemcpy(out_h, out_d, NF*pr_size, hipMemcpyDeviceToHost) ;
        hipMemcpy(u_h, u_d, AREA*Nz*pr_size, hipMemcpyDeviceToHost) ;
    
        dataCopyBackEnd = getTime();     
        dataCopyBackTotal = dataCopyBackEnd - dataCopyBackStart;
        dataCopyTotal = dataCopyInitTotal + dataCopyBtwTotal + dataCopyBackTotal;

        endTime = getTime();
        totalTime = (double) (endTime-startTime);
       

        writeBinaryDataToFile(u_h,getOutputFileName(cudaStr,"room","bin"),VOLUME);       
        writeBinaryDataToFile(out_h,getOutputFileName(cudaStr,"receiver","bin"),numberSamples);       

        // sanity check 
        for(int jj=NF-10;jj<NF;jj++)
        {
            printf("%.14lf\n",out_h[jj]);
        }

        printToString;
        printOutputs;
        writeTimingsToFile; 
	
        // Free memory
        free(si_h);free(out_h);
	hipFree(out_d);hipFree(u_d);hipFree(u1_d);
	
	exit(EXIT_SUCCESS);
}

// Room  Update Kernel
__global__ void UpDateScheme(double *u,const double* __restrict__ u1)
{
	
	// get X,Y,Z from thread and block Id's
	int X = blockIdx.x * Bx + threadIdx.x;                                              
	int Y = blockIdx.y * By + threadIdx.y;
	int Z = blockIdx.z * Bz + threadIdx.z + 1;
	
	// Test that not at halo, Z block excludes Z halo
	if( (X>0) && (X<(Nx-1)) && (Y>0) && (Y<(Ny-1)) ){
		// get linear position
		
		int cp   = Z*AREA+(Y*Nx+X);
		// local variables
		double cf  = 1.0;
		double cf2 = 1.0;
		
		int K    = (0||(X-1)) + (0||(X-(Nx-2))) + (0||(Y-1)) + (0||(Y-(Ny-2))) + (0||(Z-1)) + (0||(Z-(Nz-2)));
		
		// set loss coeffs at walls
		if(K<6){
			cf   = cf_d[0].loss1;
			cf2  = cf_d[0].loss2;
		}
        
		// Get sum of neighbours
        double S   = u1[cp-1]+u1[cp+1]+u1[cp-Nx]+u1[cp+Nx]+u1[cp-AREA]+u1[cp+AREA];
        
        // Calc update
        u[cp]    = cf*( (2.0-K*cf_d[0].l2)*u1[cp] + cf_d[0].l2*S - cf2*u[cp] );
	}
	
}

// read output and sum in input
__global__ void inout(double *u,double *out,double ins,int n)
{	
	// sum in source
	u[(Sz*AREA)+(Sy*Nx+Sx)] += ins;
	
	// non-interp read out
	out[n]  = u[(Rz*AREA)+(Ry*Nx+Rx)];
	
}

